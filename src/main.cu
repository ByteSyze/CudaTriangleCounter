#include "hip/hip_runtime.h"
/**
*
* Copyright (C) Tyler Hackett 2016
*
* CUDA Triangle Counter
*
* A quickly-written program to determine all possible combinations of
* valid triangles from a grid, allowing for certain coordinates of the
* grid to be marked as unusable.
*
* main.cu
*
* */
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "main.h"

__global__ void countTriangles(uint2 *validPoints, int *count)
{
	/* Only allow operations on blocks where x < y < z, to prevent repeat triangles*/
	if (blockIdx.x > blockIdx.y || blockIdx.y > blockIdx.z || blockIdx.x > blockIdx.z)
		return;

	uint2 x, y, z;
	x = validPoints[blockIdx.x];
	y = validPoints[blockIdx.y];
	z = validPoints[blockIdx.z];

	/*Check if the points are coplanar.*/
	if ((x.x == y.x || x.y == y.y) && (y.x == z.x || y.y == z.y) && (x.x == z.x || x.y == z.y))
		return;
	/*Check for any coincident points.*/
	if ((x.x == y.x && x.y == y.y) || (y.x == z.x && y.y == z.y) || (x.x == z.x && x.y == z.y))
		return;

	/*If the thread makes it this far, then we have a triangle that obeys the laws of geometry!*/
	atomicAdd(count, 1);
}

int main()
{
	int i = 0;
	int h_count = 0;

	int *h_countPtr = &h_count;
	int *d_count;

	//Calculate the size of the array of valid points.
	size_t size = (GRID_HEIGHT*GRID_WIDTH*sizeof(uint2) - sizeof(h_invalidPoints));
	dim3 gridSize(size / sizeof(uint2), size / sizeof(uint2), size / sizeof(uint2));

	uint2 *h_validPoints = (uint2 *)malloc(size);
	uint2 *d_validPoints;

	hipMalloc((void **)&d_validPoints, size);
	hipMalloc((void **)&d_count, sizeof(int));

	//Generate an array of all valid points within the bounds defined by GRID_WIDTH and GRID_HEIGHT.
	for (unsigned int x = 0; x < GRID_WIDTH; x++)
	{
		for (unsigned int y = 0; y < GRID_HEIGHT; y++)
		{
			uint2 p = { x, y };

			if (!isInvalidPoint(p))
			{
				h_validPoints[i] = p;
				i += 1;
			}
		}
	}

	printf("%d Valid Points.\n", i);

	/*Copy the valid points in the grid over to the device.*/
	hipMemcpy(d_validPoints, h_validPoints, size, hipMemcpyHostToDevice);
	hipMemcpy(d_count, h_countPtr, size, hipMemcpyHostToDevice);

	countTriangles<<<gridSize, 1>>>(d_validPoints, d_count);

	/*Wait for the kernel to finish*/
	hipDeviceSynchronize();

	/*Let's see how many triangles we found!*/
	hipMemcpy(h_countPtr, d_count, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d valid triangles.\n", h_count);

	free(h_validPoints);

	hipFree(d_validPoints);
	hipFree(d_count);

    return 0;
}

/**
*
* Checks h_invalidPoints for a corresponding point, represented as a uint2.
*
* */
bool isInvalidPoint(uint2 p)
{
	for each(uint2 point in h_invalidPoints)
	{
		if (point.x == p.x && point.y == p.y)
			return true;
	}
	return false;
}

