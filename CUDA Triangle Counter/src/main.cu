/**
*
* Copyright (C) Tyler Hackett 2016
*
* CUDA Triangle Counter
*
* A quickly-written program to determine all possible combinations of
* valid triangles from a grid, allowing for certain coordinates of the
* grid to be marked as unusable.
*
* main.cu
*
* */
#include "hip/hip_runtime.h"


#include <stdio.h>

int main()
{
    return 0;
}
